#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#include "math.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <chrono>
#include <ctime>
#include <iostream>
using namespace std;

hipError_t addWithCuda(int width, int height, int bpp, double* kernel, unsigned char* rgb_image, unsigned char* image_out);

using std::chrono::high_resolution_clock;

__global__ void addKernel(double* kernel, unsigned char* rgb_image, unsigned char* image_out, int* width, int* height)
{

    int x = threadIdx.x;
	int y = blockIdx.x;
	//int width = blockDim.x;
	//printf("%d %d \n", x, y);
	//printf("%d,%d \n", *height, *width);

	if (y < *height && x < *width) {
		//printf("%d %d \n", x, y);
		for (int c = 0; c < 3; c++) {
			image_out[(y* *width + x) * 3 + c] = 0.0f;
			double out = 0.0;
			for (int ky = -1; ky <= 1; ky++) {
				for (int kx = -1; kx <= 1; kx++) {
					out += rgb_image[((y + ky)* *width + (x + kx)) * 3 + c] * kernel[(ky + 1) * 3 + (kx + 1)];
				}
			}
			if (out > 255.0) {
				out = 255.0;
			}
			if (out < 0.0) {
				out = 0.0;
			}
			image_out[(y* *width + x) * 3 + c] = out;
		}
	}
    
}

int main()
{
	double mask[3][3] = { 0.0, -1.0, 0.0,
					-1.0, 5.0, -1.0,
					 0.0, -1.0, 0.0 };

	double* kernel = (double*)malloc(9*sizeof(double));
	for (int ky = 0; ky <= 2; ky++) {
		for (int kx = 0; kx <= 2; kx++) {
			kernel[ky*3 + kx] = mask[ky][kx];
		}
	}

	int width, height, bpp;
	unsigned char* rgb_image = stbi_load("test2.png", &width, &height, &bpp, 0);
	unsigned char* image_out = (unsigned char*)malloc(width * height * bpp * sizeof(unsigned char));

    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

	//hipDeviceSetLimit(hipLimitMallocHeapSize, int width*height*bpp);

	const size_t malloc_limit = size_t(2048) * size_t(2048) * size_t(2048);
	hipDeviceSetLimit(hipLimitMallocHeapSize, malloc_limit);


	auto timestart = high_resolution_clock::now();
    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(width, height, bpp, kernel, rgb_image, image_out);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	
	auto timeend = high_resolution_clock::now();
	std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(timeend - timestart).count();

	stbi_write_png("imageOut.png", width, height, 3, image_out, width*bpp);

	stbi_image_free(rgb_image);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int width, int height, int bpp, double* kernel, unsigned char* rgb_image, unsigned char* image_out)
{
	double* dev_kernel = 0;
	unsigned char* dev_rgb_image = 0;
	unsigned char* dev_image_out = 0;
	int* dev_width = 0;
	int* dev_height = 0;

    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }


    // Allocate GPU buffers for three vectors (two input, one output)    .

	cudaStatus = hipMalloc((void**)&dev_width, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_width, &width, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_height, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_height, &height, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    cudaStatus = hipMalloc((void**)&dev_kernel, 9 * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_kernel, kernel, 9 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    cudaStatus = hipMalloc((void**)&dev_rgb_image, width * height * bpp * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_image_out, width * height * bpp * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_rgb_image, rgb_image, width * height * bpp * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	dim3 blocksPerGrid(height, 1, 1);
	dim3 threadsPerBlock(width, 1, 1);

    addKernel<<<blocksPerGrid, threadsPerBlock>>>(dev_kernel, dev_rgb_image, dev_image_out, dev_width, dev_height);


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(image_out, dev_image_out, height * width * bpp * sizeof(unsigned char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
       fprintf(stderr, "hipMemcpy failed!");
        goto Error;
   }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
